#include "cudaUtils.hpp"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""

template<>
void reportCudaError<hipError_t>(hipError_t errorCode)
{
	if(errorCode != hipSuccess)
		std::cerr << hipGetErrorString(errorCode) << "\n";
}

void CudaDeleter::operator()(void* ptr) const
{
	// lol, broken on VS2015
	//static_assert(std::is_trivially_destructible<T>::value, "must be trivially destructible");
	hipFree(ptr);
}

namespace detail
{
	void* cudaCalloc(std::size_t size, std::size_t count)
	{
		void* untyped;
		hipError_t cudaStatus;
		cudaStatus = hipMalloc(&untyped, count * size);
		reportCudaError(cudaStatus);
		cudaStatus = hipMemset(untyped, 0, count * size);
		reportCudaError(cudaStatus);
		return untyped;
	}

	void copyToDevice(void* dest, const void* src, std::size_t size)
	{
		reportCudaError(hipMemcpy(dest, src, size, hipMemcpyHostToDevice));
	}

	void copyToHost(void* dest, const void* src, std::size_t size)
	{
		reportCudaError(hipMemcpy(dest, src, size, hipMemcpyDeviceToHost));
	}

	void cudaZeroOut(void* what, std::size_t size)
	{
		reportCudaError(hipMemset(what, 0, size));
	}
}

void printComputeCapability()
{
	int nDevices;
	hipGetDeviceCount(&nDevices);
	for(int i = 0; i < nDevices; ++i)
	{
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		std::cout << "Device " << i << " compute capability: " << prop.major << "." << prop.minor << "\n";
	}
}