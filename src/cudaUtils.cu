#include "cudaUtils.hpp"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""

template<>
void reportCudaError<hipError_t>(hipError_t errorCode)
{
	if(errorCode != hipSuccess)
		std::cerr << hipGetErrorString(errorCode) << "\n";
}

void CudaDeleter::operator()(void* ptr) const
{
	// lol, broken on VS2015
	//static_assert(std::is_trivially_destructible<T>::value, "must be trivially destructible");
	hipFree(ptr);
}

namespace detail
{
	void* cudaCalloc(std::size_t size, std::size_t count)
	{
		void* untyped;
		hipError_t cudaStatus;
		cudaStatus = hipMalloc(&untyped, count * size);
		reportCudaError(cudaStatus);
		cudaStatus = hipMemset(untyped, 0, count * size);
		reportCudaError(cudaStatus);
		return untyped;
	}

	void copyToDevice(void* dest, const void* src, std::size_t size)
	{
		reportCudaError(hipMemcpy(dest, src, size, hipMemcpyHostToDevice));
	}

	void copyToHost(void* dest, const void* src, std::size_t size)
	{
		reportCudaError(hipMemcpy(dest, src, size, hipMemcpyDeviceToHost));
	}

	void copyToDeviceAsync(void* dest, const void* src, std::size_t size, void* stream)
	{
		reportCudaError(hipMemcpyAsync(dest, src, size, hipMemcpyHostToDevice, static_cast<hipStream_t>(stream)));
	}

	void copyToHostAsync(void* dest, const void* src, std::size_t size, void* stream)
	{
		reportCudaError(hipMemcpyAsync(dest, src, size, hipMemcpyDeviceToHost, static_cast<hipStream_t>(stream)));
	}

	void cudaZeroOut(void* what, std::size_t size)
	{
		reportCudaError(hipMemset(what, 0, size));
	}

	void cudaZeroOutAsync(void* what, std::size_t size, void* stream)
	{
		reportCudaError(hipMemsetAsync(what, 0, size, static_cast<hipStream_t>(stream)));
	}
}

void printGpuInfo()
{
	int nDevices;
	hipGetDeviceCount(&nDevices);
	for(int i = 0; i < nDevices; ++i)
	{
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		std::cout << "Device " << i << " compute capability: " << prop.major << "." << prop.minor << "\n";
	}
}

void CudaStream::CudaStreamDeleter::operator()(void* stream) const
{
	auto raw_stream = static_cast<hipStream_t>(stream);
	hipStreamSynchronize(raw_stream);
	hipStreamDestroy(raw_stream);
}

CudaStream::CudaStream(void* raw)
{
	stream.reset(raw);
}

CudaStream& CudaStream::getDefault()
{
	static CudaStream stream(nullptr);
	return stream;
}

CudaStream::CudaStream()
{
	hipStream_t raw_stream;
	hipStreamCreateWithFlags(&raw_stream, hipStreamDefault);
	stream.reset(raw_stream);
}

void* CudaStream::get()
{
	return stream.get();
}

void CudaStream::wait() const
{
	hipStreamSynchronize(static_cast<hipStream_t>(stream.get()));
}
