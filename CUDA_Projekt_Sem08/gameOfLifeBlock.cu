#include "hip/hip_runtime.h"
#include "gameOfLifeBlock.hpp"
#include "config.hpp"
#include "hip/hip_runtime.h"
#include ""

const dim3 threadsPerBlock(threadsPerDimension, threadsPerDimension);
const dim3 dimensions(blockDimension / threadsPerBlock.x, blockDimension / threadsPerBlock.y);

__device__ int wrap(int in)
{
	return (in + blockDimension * 2) % blockDimension;
}

// in < 0: -1
// in >= 0 && in < wrapCounter: 0
// in >= wrapCounter: 1
__device__ int robert(int in)
{
	if(in < 0)
		return -1;
	if(in >= blockDimension)
		return 1;
	return 0;
}

// 0, 0 : 0
// 0, 1 : 0
// 0, 2 : 0
// 0, 3 : 1
// 0, 4 : 0
// 0, 5 : 0
// 0, 6 : 0
// 0, 7 : 0
// 0, 8 : 0
// 0, 9 : X
// 0, 10: X
// 0, 11: X
// 0, 12: X
// 0, 13: X
// 0, 14: X
// 0, 15: X
// 1, 0 : 0
// 1, 1 : 0
// 1, 2 : 1
// 1, 3 : 1
// 1, 4 : 0
// 1, 5 : 0
// 1, 6 : 0
// 1, 7 : 0
// 1, 8 : 0
// 1, 9 : X
// 1, 10: X
// 1, 11: X
// 1, 12: X
// 1, 13: X
// 1, 14: X
// 1, 15: X
__device__ bool rule(bool current, int neighbourCount)
{
	if(neighbourCount == 2)
		return current;
	else if(neighbourCount == 3)
		return true;
	else
		return false;
}

__global__ void nextGenerationKernel(bool* next_generation, const bool* const* surrounding, bool* out)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x >= blockDimension || y >= blockDimension)
		return;

	int neighbourCount =
		surrounding[center + robert(x - 1) * leftOrRight + robert(y - 1) * upOrDown][wrap(x - 1) + wrap(y - 1)*blockDimension] +
		surrounding[center + robert(y - 1) * upOrDown][x + wrap(y - 1)*blockDimension] +
		surrounding[center + robert(x + 1) * leftOrRight + robert(y - 1) * upOrDown][wrap(x + 1) + wrap(y - 1)*blockDimension] +

		surrounding[center + robert(x - 1) * leftOrRight][wrap(x - 1) + y*blockDimension] +
		surrounding[center + robert(x + 1) * leftOrRight][wrap(x + 1) + y*blockDimension] +

		surrounding[center + robert(x - 1) * leftOrRight + robert(y + 1) * upOrDown][wrap(x - 1) + wrap(y + 1)*blockDimension] +
		surrounding[center + robert(y + 1) * upOrDown][x + wrap(y + 1)*blockDimension] +
		surrounding[center + robert(x + 1) * leftOrRight + robert(y + 1) * upOrDown][wrap(x + 1) + wrap(y + 1)*blockDimension];

	next_generation[x + y*blockDimension] = rule(surrounding[center][x + y*blockDimension], neighbourCount);

	if(neighbourCount > 0)
	{
		out[center] = true;
		if(x == 0 && y == 0)
			out[center - leftOrRight - upOrDown] = true;
		if(y == 0)
			out[center - upOrDown] = true;
		if(x == blockDimension - 1 && y == 0)
			out[center + leftOrRight - upOrDown] = true;
		if(x == 0)
			out[center - leftOrRight] = true;
		if(x == blockDimension - 1)
			out[center + leftOrRight] = true;
		if(x == 0 && y == blockDimension - 1)
			out[center - leftOrRight + upOrDown] = true;
		if(y == blockDimension - 1)
			out[center + upOrDown] = true;
		if(x == blockDimension - 1 && y == blockDimension - 1)
			out[center + leftOrRight + upOrDown] = true;
	}
}

GameOfLifeBlock::GameOfLifeBlock() :
	central(blockDimension*blockDimension),
	next(blockDimension*blockDimension),
	borderCheck(maxNeighbourAndSelfCount),
	cudaSurrounding(maxNeighbourAndSelfCount),
	synchronized(hipMemcpyHostToHost), // hipMemcpyHostToHost means it's synchronized
	commited(true)
{

}

std::array<bool, maxNeighbourAndSelfCount> GameOfLifeBlock::bordersToHost()
{
	borderCheck.copyToHost();
	std::array<bool, maxNeighbourAndSelfCount> result;
	for(std::size_t i = 0; i < maxNeighbourAndSelfCount; ++i)
	{
		result[i] = borderCheck[i];
	}
	return result;
}

std::array<bool, maxNeighbourAndSelfCount> GameOfLifeBlock::nextGeneration(const std::array<const GameOfLifeBlock*, maxNeighbourAndSelfCount>& neighbours)
{
	if(!commited)
		return bordersToHost();

	if(synchronized == hipMemcpyHostToDevice)
	{
		central.copyToDevice();
		synchronized = hipMemcpyHostToHost;
	}
	cudaBzero(borderCheck);

	auto toDev = [&]()
	{
		for(std::size_t i = 0; i < maxNeighbourAndSelfCount; ++i)
		{
			cudaSurrounding[i] = neighbours[i]->central.getDevice();
		}
		cudaSurrounding[center] = central.getDevice();
		cudaSurrounding.copyToDevice();
	};
	toDev();
	nextGenerationKernel << < dimensions, threadsPerBlock >> > (next.getDevice(), cudaSurrounding.getDevice(), borderCheck.getDevice());
	auto result = bordersToHost();

	synchronized = hipMemcpyDeviceToHost;
	commited = false;
	return result;
}

void GameOfLifeBlock::setAt(std::size_t i, std::size_t j, bool what)
{
	if(synchronized == hipMemcpyDeviceToHost)
	{
		central.copyToHost();
		synchronized = hipMemcpyHostToHost;
	}

	central[j*blockDimension + i] = what;

	synchronized = hipMemcpyHostToDevice;
}

bool GameOfLifeBlock::getAt(std::size_t i, std::size_t j) const
{
	if(synchronized == hipMemcpyDeviceToHost)
	{
		central.copyToHost();
		synchronized = hipMemcpyHostToHost;
	}

	return central[j*blockDimension + i];
}

void GameOfLifeBlock::nextGenerationCommit()
{
	if(!commited)
		std::swap(central, next);
	commited = true;
}